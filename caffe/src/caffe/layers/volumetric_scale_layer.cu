#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ScaleForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* Wc, const Dtype* Ws, const int num, const int channels, 
    const int height, const int width, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height / channels;
    int c = (index / (height * width)) % channels;
    int h = (index / width) % height;
    int w = index % width;
    int c_idx = n * channels + c;
    int s_idx = n * height * width + h * width + w;
    top_data[index] = bottom_data[index] * Wc[c_idx] * Ws[s_idx];
  }
}

template <typename Dtype>
__global__ void ScaleBackward(const int nthreads, const Dtype* top_diff,
    const Dtype* Wc, const Dtype* Ws, const int num, const int channels, 
    const int height, const int width, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height / channels;
    int c = (index / (height * width)) % channels;
    int h = (index / width) % height;
    int w = index % width;
    int c_idx = n * channels + c;
    int s_idx = n * height * width + h * width + w;
    bottom_diff[index] = top_diff[index] * Wc[c_idx] * Ws[s_idx];
  }
}

template <typename Dtype>
__global__ void ScaleBackwardChannelWeight(const int nthreads, const Dtype* top_diff,
    const Dtype* bottom_data, const Dtype* Ws, const int num, const int channels, 
    const int height, const int width, Dtype* Wc_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    int c = index % channels;
    int dim = height * width;
    
    int top_idx = n * channels * height * width + c * height * width;
    int s_idx = n * dim;
    Wc_diff[index] = 0;
    for (int h = 0; h < height; ++h) {
      for (int w = 0; w < width; ++w) {
        Wc_diff[index] += top_diff[top_idx] * bottom_data[top_idx] * Ws[s_idx];
        top_idx++;
        s_idx++;
      }
    }
  }
}

template <typename Dtype>
__global__ void ScaleBackwardSpatialWeight(const int nthreads, const Dtype* top_diff,
    const Dtype* bottom_data, const Dtype* Wc, const int num, const int channels, 
    const int height, const int width, Dtype* Ws_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height;
    int h = (index / width) % height;
    int w = index % width;
    int dim = height * width;

    int top_idx = n * channels * height * width + h * width + w;
    int c_idx = n * channels;
    Ws_diff[index] = 0;
    for (int c = 0; c < channels; ++c) {
      Ws_diff[index] += top_diff[top_idx] * bottom_data[top_idx] * Wc[c_idx];
      top_idx += dim;
      c_idx++;
    }
  }
}

template <typename Dtype>
void VolumetricScaleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = top[0]->count();
  int num = top[0]->num();
  int channels = top[0]->shape(1);
  int height = top[0]->shape(2);
  int width = top[0]->shape(3);
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* Wc = bottom[1]->gpu_data();
  const Dtype* Ws = bottom[2]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  ScaleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, Wc, Ws, num, channels, height, width, top_data);
}

template <typename Dtype>
void VolumetricScaleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  int count = top[0]->count();
  int num = top[0]->num();
  int channels = top[0]->shape(1);
  int height = top[0]->shape(2);
  int width = top[0]->shape(3);
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* Wc = bottom[1]->gpu_data();
  const Dtype* Ws = bottom[2]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* Wc_diff = bottom[1]->mutable_gpu_diff();
  Dtype* Ws_diff = bottom[2]->mutable_gpu_diff();

  ScaleBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, Wc, Ws, num, channels, height, width, bottom_diff);

  ScaleBackwardSpatialWeight<Dtype><<<CAFFE_GET_BLOCKS(bottom[2]->count()), 
      CAFFE_CUDA_NUM_THREADS>>>(bottom[2]->count(), top_diff, bottom_data, Wc, 
          num, channels, height, width, Ws_diff);

  ScaleBackwardChannelWeight<Dtype><<<CAFFE_GET_BLOCKS(bottom[1]->count()), 
      CAFFE_CUDA_NUM_THREADS>>>(bottom[1]->count(), top_diff, bottom_data, Ws, 
          num, channels, height, width, Wc_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(VolumetricScaleLayer);

}  // namespace caffe

#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SumSqForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, 
    const int height, const int width, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height ;
    int h = (index / width) % height;
    int w = index % width;
    int dim = height * width;

    int bottom_index = n*width*height*channels + h*width + w;
    top_data[index]=0;
    for (int c=0; c< channels; c++){
	top_data[index] += bottom_data[bottom_index]*bottom_data[bottom_index];
	bottom_index += dim;
    }
  }
}

    
template <typename Dtype>
__global__ void SumSqBackward(const int nthreads, const Dtype* top_diff,
    const Dtype* bottom_data, const int num, const int channels, 
    const int height, const int width, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height / channels;
    int c = (index / width / height) % channels;
    int h = (index / width) % height;
    int w = index % width;
    int dim = height * width;

    int top_index = n*width*height + h*width + w;
    bottom_diff[index] = top_diff[top_index] * 2.0*bottom_data[index];
  }
}



template <typename Dtype>
void SumSqLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    // net_->ForwardPrefilled();
    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int height = top[0]->height();
    int width = top[0]->width();

    const Dtype* X = bottom[0]->gpu_data();
    Dtype* Y = top[0]->mutable_gpu_data();

    SumSqForward<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>(
        num*height*width, X, num, channels,
        height, width, Y);
    

}

template <typename Dtype>
void SumSqLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    int count = bottom[0]->count();
    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int height = bottom[0]->height();
    int width = bottom[0]->width();

    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();

    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
 
    SumSqBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, num, channels,
        height, width, bottom_diff);

}

INSTANTIATE_LAYER_GPU_FUNCS(SumSqLayer);

}

#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ScaleForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* scale_data, const int num, const int channels, 
    const int height, const int width, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height / channels;
    int h = (index / width) % height;
    int w = index % width;
    int dim = height * width;
    int scale_idx = n * dim + h * width + w;
    top_data[index] = bottom_data[index] * scale_data[scale_idx];
  }
}

template <typename Dtype>
__global__ void ScaleBackwardData(const int nthreads, const Dtype* top_diff,
    const Dtype* scale_data, const int num, const int channels, 
    const int height, const int width, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height / channels;
    int h = (index / width) % height;
    int w = index % width;
    int dim = height * width;
    int scale_idx = n * dim + h * width + w;
    bottom_diff[index] = top_diff[index] * scale_data[scale_idx];
  }
}

template <typename Dtype>
__global__ void ScaleBackwardScale(const int nthreads, const Dtype* top_diff,
    const Dtype* bottom_data, const int num, const int channels, 
    const int height, const int width, Dtype* scale_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height;
    int h = (index / width) % height;
    int w = index % width;
    int dim = height * width;

    int top_idx = n * channels * height * width + h * width + w;
    scale_diff[index] = 0;
    for (int c = 0; c < channels; ++c) {
      scale_diff[index] += top_diff[top_idx] * bottom_data[top_idx];
      top_idx += dim;
    }
  }
}

template <typename Dtype>
void ScaleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  if (scale_by_param_) {
    for (int i = 0; i < top.size(); ++i) {
      caffe_gpu_scale(top[i]->count(), factor_, bottom[i]->gpu_data(), 
        top[i]->mutable_gpu_data());
    }
  }
  else { 
    int count = top[0]->count();
    int channels = top[0]->channels();
    int height = top[0]->height();
    int width = top[0]->width();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* scale_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    
   ScaleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, scale_data, bottom[0]->num(), channels,
        height, width, top_data);
    
    // net_->ForwardPrefilled();
    /*
    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int dim = bottom[0]->count(2);
    const Dtype* X = bottom[0]->gpu_data();
    const Dtype* W = bottom[1]->gpu_data();
    Dtype* Y = top[0]->mutable_gpu_data();
    for (int i = 0; i < num; ++i) {
      for (int j = 0; j < channels; ++j) {
        caffe_gpu_mul(dim, X, W, Y);
        Y += top[0]->count(2);
        X += bottom[0]->count(2);
      }
      W += dim;
    }
    */
  }
}

template <typename Dtype>
void ScaleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (scale_by_param_) {
    for (int i = 0; i < top.size(); ++i) {
      caffe_gpu_scale(top[i]->count(), factor_, top[i]->gpu_diff(), 
        bottom[i]->mutable_gpu_diff());
    } 
  }
  else {
    int count = top[0]->count();
    int num = top[0]->num();
    int channels = top[0]->channels();
    int height = top[0]->height();
    int width = top[0]->width();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* scale_data = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype* scale_diff = bottom[1]->mutable_gpu_diff();

    ScaleBackwardData<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, scale_data, num, channels,
        height, width, bottom_diff);

    ScaleBackwardScale<Dtype><<<CAFFE_GET_BLOCKS(num * height * width), 
        CAFFE_CUDA_NUM_THREADS>>>(
          num * height * width, top_diff, bottom_data, num, channels,
          height, width, scale_diff);

    //net_->Backward();
    /*
    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int dim = bottom[0]->count(2);
    const Dtype* X = bottom[0]->gpu_data();
    const Dtype* W = bottom[1]->gpu_data();
    const Dtype* Y_diff = top[0]->gpu_diff();
    Dtype* X_diff = bottom[0]->mutable_gpu_diff();
    Dtype* W_diff = bottom[1]->mutable_gpu_diff();
    Dtype* Y_diff_X = Y_diff_X_.mutable_gpu_data();
    caffe_gpu_mul(Y_diff_X_.count(), Y_diff, X, Y_diff_X);
    caffe_gpu_set(bottom[1]->count(), (Dtype)0., W_diff);
    for (int i = 0; i < num; ++i) {
      for (int j = 0; j < channels; ++j) {
        caffe_gpu_mul(dim, Y_diff, W, X_diff);
        caffe_gpu_add(dim, Y_diff_X, W_diff, W_diff);
        
        X_diff += dim;
        Y_diff += dim;
        Y_diff_X += dim;
      }
      X += bottom[0]->count(1);
      W += bottom[1]->count(1);
      W_diff += dim;
    }
    */
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ScaleLayer);

}  // namespace caffe

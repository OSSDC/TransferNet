#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void FilterSumForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* scale_data, const int num, const int channels, 
    const int height, const int width, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height ;
    int h = (index / width) % height;
    int w = index % width;
    int dim = height * width;

    int bottom_index = n*width*height*channels + h*width + w;
    int scale_index = n*channels;
    top_data[index]=0;
    for (int c=0; c< channels; c++){
	top_data[index] += bottom_data[bottom_index]*scale_data[scale_index];
	bottom_index += dim;
	scale_index +=1;
    }
  }
}

    
template <typename Dtype>
__global__ void FilterSumBackwardData(const int nthreads, const Dtype* top_diff,
    const Dtype* scale_data, const int num, const int channels, 
    const int height, const int width, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height / channels;
    int c = (index / width / height) % channels;
    int h = (index / width) % height;
    int w = index % width;
    int dim = height * width;

    int top_index = n*width*height + h*width + w;
    int scale_index = n*channels + c;
    bottom_diff[index] = top_diff[top_index] * scale_data[scale_index];
  }
}

template <typename Dtype>
__global__ void FilterSumBackwardScale(const int nthreads, const Dtype* top_diff,
    const Dtype* bottom_data, const int num, const int channels, 
    const int height, const int width, Dtype* scale_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    int c = index % channels;
    int dim = height * width;

    int top_index = n*width*height;
    int bottom_index = n*width*height*channels + c*height*width;
    scale_diff[index]=0;
    for (int i=0; i <dim; i++){
	scale_diff[index] += top_diff[top_index] * bottom_data[bottom_index];
        top_index += 1;
        bottom_index += 1; 
    }
  }
}

template <typename Dtype>
void FilterSumLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    // net_->ForwardPrefilled();
    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int height = top[0]->height();
    int width = top[0]->width();

    const Dtype* X = bottom[0]->gpu_data();
    const Dtype* W = bottom[1]->gpu_data();
    Dtype* Y = top[0]->mutable_gpu_data();

    FilterSumForward<Dtype><<<CAFFE_GET_BLOCKS(num*height*width), CAFFE_CUDA_NUM_THREADS>>>(
        num*height*width, X, W, bottom[0]->num(), channels,
        height, width, Y);
    

    /*
    Dtype* Y = top[0]->mutable_gpu_data();
    caffe_gpu_set(top[0]->count(), (Dtype)0., Y);
    Dtype* tmp = tmp_.mutable_gpu_data();

    for (int i = 0; i < num; ++i) {
      for (int j = 0; j < channels; ++j) {
        caffe_gpu_scale(dim, W[j], X, tmp);
        caffe_gpu_add(dim, tmp, Y, Y);
        X += bottom[0]->count(2);
      }
      Y += dim;
      W += channels;
    }
   */
}

template <typename Dtype>
void FilterSumLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    int count = bottom[0]->count();
    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int height = bottom[0]->height();
    int width = bottom[0]->width();

    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* scale_data = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();

    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype* scale_diff = bottom[1]->mutable_gpu_diff();
 
    FilterSumBackwardData<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, scale_data, num, channels,
        height, width, bottom_diff);

   
    FilterSumBackwardScale<Dtype><<<CAFFE_GET_BLOCKS(num*channels), CAFFE_CUDA_NUM_THREADS>>>(
        num*channels, top_diff, bottom_data, num, channels,
        height, width, scale_diff);


/*
    Dtype* Y_diff_X = Y_diff_X_.mutable_gpu_data();
    Dtype* tmp = tmp_.mutable_gpu_data();
    caffe_gpu_set(bottom[1]->count(), (Dtype)0., W_diff);

    for (int i = 0; i < num; ++i) {
      for (int j = 0; j < channels; ++j) {
        caffe_gpu_scale(dim, W[j], Y_diff, X_diff);
        caffe_gpu_mul(dim, Y_diff, X, Y_diff_X);
	caffe_gpu_asum(dim, Y_diff_X, W_diff); //very suspicous

        X_diff += dim;
        Y_diff_X += dim;
        W_diff += 1;
        X += dim;
      }
      Y_diff += dim;
      W += channels;
    }

*/
}

INSTANTIATE_LAYER_GPU_FUNCS(FilterSumLayer);

}

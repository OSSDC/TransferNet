#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ChannelSumForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* scale_data, const int num, const int channels, 
    const int height, const int width, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels ;
    int c = index % channels;
    int dim = height * width;

    int bottom_index = n*width*height*channels + c*width*height;
    top_data[index]=0;
    for (int c=0; c< dim; ++c){
	top_data[index] += bottom_data[bottom_index]*scale_data[index];
	bottom_index += 1;
    }
  }
}

    
template <typename Dtype>
__global__ void ChannelSumBackwardData(const int nthreads, const Dtype* top_diff,
    const Dtype* scale_data, const int num, const int channels, 
    const int height, const int width, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / width / height / channels;
    int c = (index / width / height) % channels;

    int w_index = n*channels + c;
    bottom_diff[index] = top_diff[w_index] * scale_data[w_index];
  }
}


template <typename Dtype>
__global__ void ChannelSumBackwardScale(const int nthreads, const Dtype* top_diff,
    const Dtype* bottom_data, const int num, const int channels, 
    const int height, const int width, Dtype* scale_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    int c = index % channels;
    int dim = height * width;

    int bottom_index = n*width*height*channels + c*height*width;
    scale_diff[index]=0;
    for (int i=0; i <dim; i++){
	scale_diff[index] += top_diff[index] * bottom_data[bottom_index];
        bottom_index += 1; 
    }
  }
}

template <typename Dtype>
void ChannelSumLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    // net_->ForwardPrefilled();
    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int height = bottom[0]->height();
    int width = bottom[0]->width();

    const Dtype* X = bottom[0]->gpu_data();
    const Dtype* W = bottom[1]->gpu_data();
    Dtype* Y = top[0]->mutable_gpu_data();

    ChannelSumForward<Dtype><<<CAFFE_GET_BLOCKS(num*channels), CAFFE_CUDA_NUM_THREADS>>>(
        num*channels, X, W, bottom[0]->num(), channels,
        height, width, Y);
    

    /*
    Dtype* Y = top[0]->mutable_gpu_data();
    caffe_gpu_set(top[0]->count(), (Dtype)0., Y);
    Dtype* tmp = tmp_.mutable_gpu_data();

    for (int i = 0; i < num; ++i) {
      for (int j = 0; j < channels; ++j) {
        caffe_gpu_scale(dim, W[j], X, tmp);
        caffe_gpu_add(dim, tmp, Y, Y);
        X += bottom[0]->count(2);
      }
      Y += dim;
      W += channels;
    }
   */
}

template <typename Dtype>
void ChannelSumLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    int count = bottom[0]->count();
    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int height = bottom[0]->height();
    int width = bottom[0]->width();

    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* scale_data = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();

    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype* scale_diff = bottom[1]->mutable_gpu_diff();
 
    ChannelSumBackwardData<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, scale_data, num, channels,
        height, width, bottom_diff);

   
    ChannelSumBackwardScale<Dtype><<<CAFFE_GET_BLOCKS(num*channels), CAFFE_CUDA_NUM_THREADS>>>(
        num*channels, top_diff, bottom_data, num, channels,
        height, width, scale_diff);


/*
    Dtype* Y_diff_X = Y_diff_X_.mutable_gpu_data();
    Dtype* tmp = tmp_.mutable_gpu_data();
    caffe_gpu_set(bottom[1]->count(), (Dtype)0., W_diff);

    for (int i = 0; i < num; ++i) {
      for (int j = 0; j < channels; ++j) {
        caffe_gpu_scale(dim, W[j], Y_diff, X_diff);
        caffe_gpu_mul(dim, Y_diff, X, Y_diff_X);
	caffe_gpu_asum(dim, Y_diff_X, W_diff); //very suspicous

        X_diff += dim;
        Y_diff_X += dim;
        W_diff += 1;
        X += dim;
      }
      Y_diff += dim;
      W += channels;
    }

*/
}

INSTANTIATE_LAYER_GPU_FUNCS(ChannelSumLayer);

}

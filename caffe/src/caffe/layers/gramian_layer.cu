#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template<typename Dtype>
__global__ void GramianTopDiff(const int nthreads, const Dtype* top_diff, const Dtype* bottom_data, 
    const int D, const int M, Dtype* bottom_diff) {
  // dim = M*M;
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / D / M;
    int k = (index / M) % D;
    int i = index % M;

    int bottom_diff_idx = n*D*M + k*M + i;
    int bottom_data_idx = n*D*M + k*M;
    int top_diff_idx = n*M*M;
    bottom_diff[bottom_diff_idx]=0;
    for (int j=0; j< M; ++j){
	bottom_diff[bottom_diff_idx] += bottom_data[bottom_data_idx+j]*(top_diff[top_diff_idx+i*M+j] + top_diff[top_diff_idx+j*M+i]);
    }
    
  }
}



template <typename Dtype>
void GramianLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
//    Forward_cpu(bottom, top);

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();    

    int num = bottom[0]->shape(0);
    int channels = bottom[0]->shape(1);
    int height = bottom[0]->height();
    int width = bottom[0]->width();
 

    int M_ = height*width;
    int K_ = channels;
    int N_ = M_;

    for (int i=0; i< num; ++i){
        caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, M_, N_, K_, (Dtype)1.,
          bottom_data, bottom_data, (Dtype)0., top_data);
        bottom_data += M_*K_;
        top_data += M_*M_;
    }

}


template <typename Dtype>
void GramianLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
//    Backward_cpu(top, propagate_down, bottom);
    const Dtype* bottom_data = bottom[0]->gpu_data();    
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

    int channels = bottom[0]->shape(1);
    int height = bottom[0]->height();
    int width = bottom[0]->width();

    int M_ = height*width;
    int K_ = channels;
    int N_ = M_;

    int count = bottom[0]->count();

    GramianTopDiff<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data,  K_, M_, bottom_diff);

/*
    int count = top[0]->count();
    Dtype* top_diff_A = A_b.mutable_gpu_data();

    GramianTopDiff_A<Dtype><<<CAFFE_GET_BLOCKS(num*M_*M_), CAFFE_CUDA_NUM_THREADS>>>(
        num*M_*M_, top_diff, num, M_*M_, M_, top_diff_A);


    
    caffe_gpu_set<Dtype>(count, (Dtype)0., A_b_data);
    caffe_gpu_add<Dtype>(count, top_diff, A_b_data, A_b_data);
    for (int i=0; i< num; ++i){
	for( int j=0; j< M_; j++){
	    A_b_data[j*M_+j] += top_diff[j*M_+j];
	    LOG(INFO) << "copied_data " << A_b_data[j*M_+j] << "original_data " << top_diff[j*M_+j];
	    std::cout << "copied_data " << A_b_data[j*M_+j] << "original_data " << top_diff[j*M_+j] << std::endl;
	}
  	A_b_data += M_*M_;
        top_diff += M_*M_;
    }    
    
    A_b_data = A_b.mutable_gpu_data();
    
    
    for (int i=0; i< num; ++i){
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, K_, M_, M_,  (Dtype)1.,
          bottom_data, top_diff_A, (Dtype)0., bottom_diff);
	bottom_data += M_*K_;
        top_diff_A += M_*M_;
	bottom_diff += M_*K_;
    }
    */
/*
    for (int i=0; i< num; ++i){
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, K_, M_, M_,  (Dtype)2.,
          bottom_data, top_diff, (Dtype)0., bottom_diff);
	bottom_data += M_*K_;
        top_diff += M_*M_;
	bottom_diff += M_*K_;
    }
*/
}
INSTANTIATE_LAYER_GPU_FUNCS(GramianLayer);

}
